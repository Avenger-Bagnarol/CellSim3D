#include "hip/hip_runtime.h"
//#define FORCE_DEBUG
//#define PRINT_VOLUMES
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <locale.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <streambuf>
#include <cstring>
#include <string>

#include <hip/hip_runtime.h>
//#include "hip/hip_runtime_api.h"
#include "postscript.h"
#include "marsaglia.h"
#include "IntegrationKernels.h"
#include "RandomVector.h"

#include "json/json.h"

#define CudaErrorCheck() { \
      hipError_t e = hipGetLastError(); \
      if (e!=hipSuccess){\
          printf("Cuda failure %s: %d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
          exit(0); \
      }            \
    }

float mass;                                        //  M
float repulsion_range,    attraction_range;        //  LL1, LL2
float repulsion_strength, attraction_strength;     //  ST1, ST2

// variables to allow for different stiffnesses
float stiffness1;
float stiffness2; 
float* d_Youngs_mod;
float* youngsModArray; 
bool useDifferentStiffnesses;
float softYoungsMod;
int numberOfSofterCells;
bool duringGrowth;
bool daughtSameStiffness;
float closenessToCenter;
int startAtPop;

bool chooseRandomCellIndices;
float fractionOfSofterCells;

float viscotic_damping, internal_damping;          //  C, DMP
float gamma_visc;
float zOffset; // Offset from Z = 0 for starting positions.
int ranZOffset;
int   Time_steps;
float divVol;
float delta_t;
int   Restart;
int   trajWriteInt; // trajectory write interval
int   countOnlyInternal; // 0 - Count all new cells
                         // 1 - Count only the cells born within 0.6Rmax from
                         //     the center of mass of the system
float radFrac; // The factor to count cells within a raduys (<Rmax)

int   overWriteMitInd; // 0 No, 1 yes

int newCellCountInt; // Interval at which to count the divided cells
int equiStepCount;
const char* ptrajFileName;
char trajFileName[256];

// equilibrium length of springs between fullerene atoms
float R0  = 0.13517879937327418f;

float L1  = 3.0f;       // the initial fullerenes are placed in
// an X x Y grid of sizne L1 x L1


// the three nearest neighbours of C180 atoms
int   C180_nn[3*192];
int   C180_sign[180];
// device: the three nearest neighbours of C180 atoms
int   *d_C180_nn;
int   *d_C180_sign;

int   CCI[2][271];       // list of nearest neighbor carbon pairs in the fullerne
// number of pairs = 270

int   C180_56[92*7];     // 12 lists of atoms forming pentagons 1 2 3 4 5 1 1 and
// 80 lists of atoms forming hexagons  1 2 3 4 5 6 1
int   *d_C180_56;

float *d_volume;
float *volume;
char* cell_div;
char* d_cell_div;
int num_cell_div;
int* cell_div_inds;

char mitIndFileName[256]; 

float *d_pressList;
float *pressList;
int* d_resetIndices;
int* resetIndices; 


float* d_velListX; 
float* d_velListY; 
float* d_velListZ;

float* d_velHtsX;
float* d_velHtsY;
float* d_velHtsZ;

float* velListX; 
float* velListY; 
float* velListZ; 

// Params related to population modelling
int doPopModel;
char* didCellDie;
float totalFood;
float* d_totalFood;
int haylimit;
int cellLifeTime;
float cellFoodCons; // baseline food consumption
float cellFoodConsDiv; // Extra good consumption when cell divides
float cellFoodRel; // Food released when cell dies (should < total consumed food)
float maxPressure;
float minPressure;
float rMax;
float maxPop; 

// Params related to having walls in the simulation
int useWalls;
char perpAxis[2];
float threshDist;
float dAxis;
float wallLen;
float wallWidth;
float wall1, wall2;
float wallWStart, wallWEnd;
float wallLStart, wallLEnd;

float boxLength, boxMin[3];
bool useRigidSimulationBox; 
float* d_boxMin;

int No_of_threads; // ie number of staring cells
int Side_length;
int ex, ey;


float  *X,  *Y,  *Z;     // host: atom positions

float *d_XP, *d_YP, *d_ZP;     // device: time propagated atom positions
float  *d_X,  *d_Y,  *d_Z;     // device: present atom positions
float *d_XM, *d_YM, *d_ZM;     // device: previous atom positions


float* d_Fx;
float* d_Fy;
float* d_Fz;

// host: minimal bounding box for fullerene
float *bounding_xyz;
float *d_bounding_xyz;   // device:  bounding_xyz

// global minimum and maximum of x and y, preprocessfirst
// global minimum and maximum of x and y, postprocesssecond
float *d_Minx, *d_Maxx, *d_Miny, *d_Maxy, *d_Minz, *d_Maxz;
float *Minx, *Maxx, *Miny, *Maxy, *Minz, *Maxz;

float DL;
int Xdiv, Ydiv, Zdiv;

int *d_NoofNNlist;
int *d_NNlist;
int *NoofNNlist;
int *NNlist;

float *d_CMx, *d_CMy, *d_CMz;
float *CMx, *CMy, *CMz;
float sysCMx = 1.0, sysCMy = 1.0, sysCMz = 1.0;
float sysCMx_old = 0.0, sysCMy_old = 0.0, sysCMz_old = 0.0;

//float Pressure;          // pressure
//float Temperature;       // equation of state relates Pressure and Temperature

int  No_of_C180s;        // the global number of C180 fullerenes
int  No_of_C180s_in;     // the number of C180s near the center of mass of the system
int MaxNoofC180s; 

float *ran2;             // host: ran2[]
float *d_ran2;           // device: ran2[], used in celldivision

int *NDIV;               // # of divisions

// Parameters related to division
bool useDivPlaneBasis;
float divPlaneBasis[3]; 

long int GPUMemory;
long int CPUMemory;


int main(int argc, char *argv[])
{
  int i;
  int globalrank,step;
  int noofblocks, threadsperblock, prevnoofblocks;
  int Orig_No_of_C180s, newcells;
  int reductionblocks;
  //float PSS;
  float s, theta, phi;
  FILE *outfile;
  FILE *trajfile; // pointer to xyz file
  hipError_t myError;

  int* dividingCells; //Cells that are about to divide
  int* totalCells; // No. of cells at every Dividing_steps


  int* num_new_cells_per_step;
  int countOffset = 0;

  //int min_no_of_cells = 10;

  printf("CellDiv version 0.9\n");

  if ( argc != 3 )
  {
      printf("Usage: CellDiv no_of_threads inpFile.json\n");
      return(0);
  }

  No_of_threads = atoi(argv[1]);

  char inpFile[256];
  strcpy(inpFile, argv[2]);

  if ( read_json_params(inpFile)          != 0 ) return(-1);

  printf("%d\n", MaxNoofC180s); 

  Side_length   = (int)( sqrt( (double)No_of_threads )+0.5);
  if ( No_of_threads > MaxNoofC180s || Side_length*Side_length != No_of_threads )
  {
      printf("Usage: Celldiv no_of_threads\n");
      printf("       no_of_threads should be a square, n^2, < %d\n", MaxNoofC180s);
      return(0);
  }


  No_of_C180s      = No_of_threads;
  Orig_No_of_C180s = No_of_C180s;
  GPUMemory = 0L;
  CPUMemory = 0L;

  //if ( read_global_params()               != 0 ) return(-1);
  if ( read_fullerene_nn()                != 0 ) return(-1);
  if ( generate_random(Orig_No_of_C180s)  != 0 ) return(-1);
  if ( initialize_C180s(Orig_No_of_C180s) != 0 ) return(-1);
  NDIV = (int *)calloc(MaxNoofC180s,sizeof(int));
  CPUMemory += MaxNoofC180s*sizeof(int);
  for ( i = 0; i < No_of_threads; ++i ) NDIV[i] = 1;
  for ( i = No_of_threads; i < MaxNoofC180s; ++i ) NDIV[i] = 0;

  // empty the psfil from previous results
  outfile = fopen("psfil","w");
  if ( outfile == NULL ) {printf("Unable to open file psfil\n");return(-1);}
  fclose(outfile);

  /* PM
     Allocate memory for the dividingCells array that will be used to
     calculate the mitotic index.
  */

  dividingCells = (int *)calloc((Time_steps/newCellCountInt), sizeof(int));
  totalCells = (int *)calloc((Time_steps/newCellCountInt), sizeof(int));
  num_new_cells_per_step = (int *)calloc(Time_steps, sizeof(int));

  CPUMemory += (2L*(long)(Time_steps/newCellCountInt) + 1L + (long)Time_steps) * sizeof(int);



  // Allocate initializing memory for didCellDie

  didCellDie = (char *)(calloc(MaxNoofC180s, sizeof(char)));
  CPUMemory += (long)MaxNoofC180s * sizeof(char);

  hipDeviceProp_t deviceProp = getDevice();

  if ( hipSuccess != hipMalloc( (void **)&d_C180_nn, 3*192*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_C180_sign, 180*sizeof(int))) return(-1);
  GPUMemory +=  3*192*sizeof(int) + 180*sizeof(int);
  //  hipError_t myError = hipGetLastError();
  //     if ( hipSuccess != myError )
  //         { printf( "1: Error %d: %s!\n",myError,hipGetErrorString(myError) );return(-1);}

  if ( hipSuccess != hipMalloc( (void **)&d_XP , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_YP , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ZP , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_X  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Y  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Z  , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_XM , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_YM , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ZM , 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_bounding_xyz , MaxNoofC180s*6*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMx ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMy ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_CMz ,          MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_volume ,       MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_cell_div ,     MaxNoofC180s*sizeof(char))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Minx ,         1024*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Maxx ,         1024*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Miny ,         1024*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Maxy ,         1024*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Minz ,         1024*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Maxz ,         1024*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_NoofNNlist ,   1024*1024*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_NNlist ,    32*1024*1024*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_C180_56,       92*7*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_ran2 , 10000*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_pressList, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_velListX, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_velListY, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_velListZ, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_velHtsX, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_velHtsY, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_velHtsZ, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_resetIndices, MaxNoofC180s*sizeof(int))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Fx, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Fy, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Fz, 192*MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_Youngs_mod, MaxNoofC180s*sizeof(float))) return(-1);
  if ( hipSuccess != hipMalloc( (void **)&d_boxMin, 3*sizeof(float))) return(-1); 
  


  bounding_xyz = (float *)calloc(MaxNoofC180s*6, sizeof(float));
  CMx   = (float *)calloc(MaxNoofC180s, sizeof(float));
  CMy   = (float *)calloc(MaxNoofC180s, sizeof(float));
  CMz   = (float *)calloc(MaxNoofC180s, sizeof(float));
  volume= (float *)calloc(MaxNoofC180s, sizeof(float));
  cell_div = (char *)calloc(MaxNoofC180s, sizeof(char));
  cell_div_inds = (int *)calloc(MaxNoofC180s, sizeof(int));
  Minx  = (float *)calloc(1024, sizeof(float));
  Maxx  = (float *)calloc(1024, sizeof(float));
  Miny  = (float *)calloc(1024, sizeof(float));
  Maxy  = (float *)calloc(1024, sizeof(float));
  Minz  = (float *)calloc(1024, sizeof(float));
  Maxz  = (float *)calloc(1024, sizeof(float));
  NoofNNlist = (int *)calloc( 1024*1024,sizeof(int));
  NNlist =  (int *)calloc(32*1024*1024, sizeof(int));
  pressList = (float *)calloc(MaxNoofC180s, sizeof(float));
  resetIndices = (int *)calloc(MaxNoofC180s, sizeof(int)); 

  CPUMemory += MaxNoofC180s*7L*sizeof(float);
  CPUMemory += MaxNoofC180s*sizeof(float);
  CPUMemory += 3L*MaxNoofC180s*sizeof(float);
  CPUMemory += 6L*1024L*sizeof(float);
  CPUMemory += MaxNoofC180s*sizeof(char);
  CPUMemory += MaxNoofC180s*sizeof(int);
  CPUMemory += MaxNoofC180s*sizeof(int); 


  hipMemcpy(d_pressList, pressList, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);

  velListX = (float *)calloc(192*MaxNoofC180s, sizeof(float)); 
  velListY = (float *)calloc(192*MaxNoofC180s, sizeof(float)); 
  velListZ = (float *)calloc(192*MaxNoofC180s, sizeof(float));

  hipMemcpy(d_velListX, velListX, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velListY, velListY, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velListZ, velListZ, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_velHtsX, velListX, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velHtsY, velListY, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_velHtsZ, velListZ, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_Fx, velListX, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Fy, velListY, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Fz, velListZ, 192*MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_volume, velListZ, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice); 

  // Set the Youngs_mod for the cells
  youngsModArray = (float *)calloc(MaxNoofC180s, sizeof(float));
  if (useDifferentStiffnesses){
      
      if (!duringGrowth){
          
          for (int i = 0; i < MaxNoofC180s; i++){
              youngsModArray[i] = stiffness1;
          }
          
      } else {
          
          if (fractionOfSofterCells > 0){
              
              int c = 0;
              for (int i = 0; i < MaxNoofC180s; i++){
                  float ran1[1];
                  ranmar(ran1, 1);
                  if (ran1[0] <= fractionOfSofterCells){
                      youngsModArray[i] = stiffness2;
                      c++; 
                  }
                  else
                      youngsModArray[i] = stiffness1;
                  
              }
              
              float fset = ((float)c)/((float)MaxNoofC180s);
              if ( abs(fset - fractionOfSofterCells) > 1e-1 )
                  printf("WARNING: %.2f %% cells set to softer, %.2f %% requested\n",
                         fset*100, fractionOfSofterCells*100);
              
          } else if (numberOfSofterCells > 0){
              
              if (!chooseRandomCellIndices){
                  printf("ERROR: Cell indices can only be chose randomly during growth\n");
                  return -11;
              }
              
              for (int i = 0; i < numberOfSofterCells; i++){
                  youngsModArray[i] = stiffness2; 
              }

               for (int i = numberOfSofterCells; i < MaxNoofC180s; i++){
                  youngsModArray[i] = stiffness1; 
              }
              
          }
          
      }
  } else if (!useDifferentStiffnesses){
      
      for (int i = 0; i < MaxNoofC180s; i++){
          youngsModArray[i] = stiffness1;
      }
  }

  for (int i = 0; i < MaxNoofC180s; i++){
      printf("cell: %d, stiffness = %f\n", i, youngsModArray[i]);
  }
  
  
  hipMemcpy(d_Youngs_mod, youngsModArray, MaxNoofC180s*sizeof(float), hipMemcpyHostToDevice);
  CudaErrorCheck();
        
  // Better way to see how much GPU memory is being used.
  size_t totalGPUMem;
  size_t freeGPUMem;

  if ( hipSuccess != hipMemGetInfo ( &freeGPUMem, &totalGPUMem ) ) {
      printf("Couldn't read GPU Memory status\nExiting...");
      exit(1);
  }

  GPUMemory = totalGPUMem - freeGPUMem;

  printf("   Total amount of GPU memory used =    %8.2lf MB\n",GPUMemory/1000000.0);
  printf("   Total amount of CPU memory used =    %8.2lf MB\n",CPUMemory/1000000.0);

  hipMemcpy(d_C180_nn,   C180_nn,   3*192*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_C180_sign, C180_sign, 180*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_C180_56,   C180_56,   7*92*sizeof(int),hipMemcpyHostToDevice);

  hipMemcpy(d_XP, X, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_YP, Y, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_ZP, Z, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_X,  X, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Y,  Y, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_Z,  Z, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_XM, X, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_YM, Y, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_ZM, Z, 192*MaxNoofC180s*sizeof(float),hipMemcpyHostToDevice);

  hipMemcpy(d_cell_div, cell_div, MaxNoofC180s*sizeof(char), hipMemcpyHostToDevice);


  prevnoofblocks  = No_of_C180s;
  noofblocks      = No_of_C180s;
  threadsperblock = 192;
  printf("   no of blocks = %d, threadsperblock = %d, no of threads = %ld\n",
         noofblocks, threadsperblock, ((long) noofblocks)*((long) threadsperblock));

  bounding_boxes<<<No_of_C180s,32>>>(No_of_C180s,d_XP,d_YP,d_ZP,d_X,d_Y,d_Z,d_XM,d_YM,d_ZM,
                                     d_bounding_xyz, d_CMx, d_CMy, d_CMz);


  reductionblocks = (No_of_C180s-1)/1024+1;
  minmaxpre<<<reductionblocks,1024>>>( No_of_C180s, d_bounding_xyz,
                                       d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
  CudaErrorCheck(); 
  minmaxpost<<<1,1024>>>(reductionblocks, d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
  CudaErrorCheck(); 
  hipMemset(d_NoofNNlist, 0, 1024*1024);
  hipMemcpy(Minx, d_Minx, 6*sizeof(float),hipMemcpyDeviceToHost);
  //  DL = 3.8f;
  DL = 2.9f;
  //DL = divVol; 
  Xdiv = (int)((Minx[1]-Minx[0])/DL+1);
  Ydiv = (int)((Minx[3]-Minx[2])/DL+1);
  Zdiv = (int)((Minx[5]-Minx[4])/DL+1);
  makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_bounding_xyz, Minx[0], Minx[2], Minx[4],
                                         attraction_range, Xdiv, Ydiv, Zdiv, d_NoofNNlist, d_NNlist, DL);
  CudaErrorCheck(); 
  globalrank = 0;

  // open trajectory file
  trajfile = fopen (trajFileName, "w");
  if ( trajfile == NULL)
  {
      printf("Failed to open %s \n", trajFileName);
      return -1;
  }

  FILE* velFile = fopen("velocity2.xyz", "w"); 

  //OpenBinaryFile("binFile.hist", &bFA, trajWriteInt);

  write_traj(0, trajfile);
 // WriteToBinaryFile(X, Y, Z,
 //                    No_of_C180s, 0, &bFA);

  // Set up walls if needed
  if (useWalls == 1){
      // First we must make sure that the walls surround the
      // starting system.
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        d_XP, d_YP, d_ZP,
                                        d_CMx, d_CMy, d_CMz);
      hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      float COMx = 0, COMy = 0, COMz = 0;

      for(int cell = 0; cell < No_of_C180s; cell++){
          COMx += CMx[cell];
          COMy += CMy[cell];
          COMz += CMz[cell];
      }

      COMx = COMx/No_of_C180s;
      COMy = COMy/No_of_C180s;
      COMz = COMz/No_of_C180s;


      if (perpAxis[0] == 'Z' || perpAxis[0] == 'z' ){
          // Check that the walls are far enough from the beginning cells
          float tempZ[192*No_of_C180s];
          hipMemcpy(tempZ, d_Z, 192*No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
          std::sort(tempZ, tempZ+No_of_C180s);
          float radius = 3.0 * divVol / 4.0;
          radius = radius/3.14159;
          radius = std::pow(radius, 0.33333333333);
          dAxis = dAxis * 2 * radius;

          if (dAxis < (tempZ[No_of_C180s] - tempZ[0])){
                  printf("Distance between walls is too small\nExiting...");
                  printf("Starting system size= %f\nWall gap = %f",
                         tempZ[No_of_C180s] - tempZ[0], dAxis);
                  return(-1);
              }

          wall1 = COMz - (dAxis/2.0);
          wall2 = COMz + (dAxis/2.0);
          wallLStart = COMx - (wallLen/2.0);
          wallLEnd = COMx + (wallLen/2.0);
          wallWStart = COMy - (wallWidth/2.0);
          wallWEnd = COMy + (wallWidth/2.0);
      }
      else {
          printf(" Invalid wall axis selection %s\nExiting...", perpAxis);
          return(-1);
      }

  }


  // Initialize pressures

  for (int cell = 0; cell < No_of_C180s; cell++){
      pressList[cell] = minPressure;
  }

  hipMemcpy(d_pressList, pressList, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);

  float rGrowth = 0;
  bool growthDone = false;

  boxMin[0] = 0;
  boxMin[1] = 0;
  boxMin[2] = 0;
  
  // Setup simulation box, if needed (non-pbc)
  if (useRigidSimulationBox){
      printf("   Setup rigid (non-PBC) box...\n"); 
      boxLength = ceil(max( (Minx[5]-Minx[4]), max( (Minx[1]-Minx[0]), (Minx[3]-Minx[2]) ) ));
      if (Side_length < 5) boxLength = boxLength * 5; 
      boxMin[0] = floor(Minx[0] - 0.1);
      boxMin[1] = floor(Minx[2] - 0.1);
      boxMin[2] = floor(Minx[4] - 0.1);
      printf("   Done!\n");
      printf("   Simulation box minima:\n   X: %f, Y: %f, Z: %f\n", boxMin[0], boxMin[1], boxMin[2]);
      printf("   Simulation box length = %f\n", boxLength);
  }

  hipMemcpy(d_boxMin, boxMin, 3*sizeof(float), hipMemcpyHostToDevice); 
  
  // Simulation loop
  for ( step = 1; step < Time_steps+1 + equiStepCount; step++)
  {
      if (doPopModel == 1){
            rGrowth = rMax * (1 - (No_of_C180s*1.0/maxPop));
            // dr = -rGrowth(a + b*rGrowth)
            // rGrowth += dr * delta_t ;
            // dN/dT = N*R
            // dR/dT = -R(a+bR)
            // 
            if (rGrowth < 0) rGrowth =0; 
      }
      else {
      rGrowth = rMax;
      }
      PressureUpdate <<<No_of_C180s/512 + 1, 512>>> (d_pressList, minPressure, maxPressure, rGrowth, No_of_C180s);
      
      if ( (step)%1000 == 0)
      {
          printf("   time %-8d %d cells, rGrowth %f, maxPop %f\n",step,No_of_C180s, rGrowth, maxPop);
      }

      noofblocks      = No_of_C180s;
      if ( prevnoofblocks < noofblocks )
      {
          prevnoofblocks  = noofblocks;
          //        printf("             no of thread blocks = %d, threadsperblock = %d, no of threads = %ld\n",
          //             noofblocks, threadsperblock, ((long) noofblocks)*((long) threadsperblock));
      }

#ifdef FORCE_DEBUG
      printf("time %d  pressure = %f\n", step, Pressure);
#endif
          //printf("\n new step \n"); 
      propagate<<<noofblocks,threadsperblock>>>( No_of_C180s, d_C180_nn, d_C180_sign,
                                                 d_XP, d_YP, d_ZP, d_X,  d_Y,  d_Z, d_XM, d_YM, d_ZM,
                                                 d_CMx, d_CMy, d_CMz,
                                                 R0, d_pressList, d_Youngs_mod , stiffness1, 
                                                 internal_damping, delta_t, d_bounding_xyz,
                                                 attraction_strength, attraction_range,
                                                 repulsion_strength, repulsion_range,
                                                 viscotic_damping, mass,
                                                 Minx[0], Minx[2], Minx[4], Xdiv, Ydiv, Zdiv, d_NoofNNlist, d_NNlist, DL, gamma_visc,
                                                 wall1, wall2,
                                                 threshDist, useWalls,
                                                 d_velListX, d_velListY, d_velListZ,
                                                 useRigidSimulationBox, boxLength, d_boxMin);
      
      CenterOfMass<<<No_of_C180s,256>>>(No_of_C180s,
                                        d_XP, d_YP, d_ZP,
                                        d_CMx, d_CMy, d_CMz);
      if (step <= Time_steps && rGrowth > 0){
        // ------------------------------ Begin Cell Division ------------------------------------------------


        volumes<<<No_of_C180s,192>>>(No_of_C180s, d_C180_56,
                                     d_XP, d_YP, d_ZP,
                                     d_CMx , d_CMy, d_CMz,
                                     d_volume, d_cell_div, divVol);

        count_and_get_div();
        for (int divCell = 0; divCell < num_cell_div; divCell++) {
          globalrank = cell_div_inds[divCell];
          float norm[3];

          if (useDivPlaneBasis)
              GetRandomVectorBasis(norm, divPlaneBasis);
          else
              GetRandomVector(norm); 

          hipMemcpy( d_ran2, norm, 3*sizeof(float), hipMemcpyHostToDevice); 
          
          NDIV[globalrank] += 1;

          cell_division<<<1,256>>>(globalrank,
                                   d_XP, d_YP, d_ZP,
                                   d_X, d_Y, d_Z,
                                   d_CMx, d_CMy, d_CMz,
                                   No_of_C180s, d_ran2, repulsion_range);
          resetIndices[divCell] = globalrank;
          resetIndices[divCell + num_cell_div] = No_of_C180s;

          if (daughtSameStiffness){
              youngsModArray[No_of_C180s] = youngsModArray[globalrank];
              hipMemcpy(d_Youngs_mod+No_of_C180s, youngsModArray+No_of_C180s,
                         sizeof(float), hipMemcpyHostToDevice);
          }

          ++No_of_C180s;
        }

        hipMemcpy(d_resetIndices, resetIndices, 2*num_cell_div*sizeof(int),
                   hipMemcpyHostToDevice); 

        PressureReset <<<(2*num_cell_div)/512 + 1, 512>>> (d_resetIndices, d_pressList, minPressure, 2*num_cell_div); 

        totalFood -= num_cell_div*cellFoodConsDiv;

        if (countOnlyInternal == 1){
          num_cell_div -= num_cells_far();
        }

        num_new_cells_per_step[step-1] = num_cell_div;
        if (step%newCellCountInt == 0){
          newcells = 0;
          for (int i = 0; i < newCellCountInt; i++) {
            newcells += num_new_cells_per_step[countOffset + i];
          }
          dividingCells[(step-1)/newCellCountInt] = newcells;
          totalCells[(step-1)/newCellCountInt] = No_of_C180s - newcells;
          // Need to make sure this is how MIs are even calculated
          countOffset += newCellCountInt;
        }
        // --------------------------------------- End Cell Division -----------
      }

      // ----------------------------------------- Begin Cell Death ------------

      // Placeholder************************************************************

      // ----------------------------------------- End Cell Death --------------

      
      bounding_boxes<<<No_of_C180s,32>>>(No_of_C180s,
                                         d_XP,d_YP,d_ZP,d_X,d_Y,d_Z,d_XM,d_YM,d_ZM,
                                         d_bounding_xyz, d_CMx, d_CMy, d_CMz);
      CudaErrorCheck();

      reductionblocks = (No_of_C180s-1)/1024+1;
      minmaxpre<<<reductionblocks,1024>>>( No_of_C180s, d_bounding_xyz,
                                           d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
      CudaErrorCheck(); 

      minmaxpost<<<1,1024>>>( reductionblocks, d_Minx, d_Maxx, d_Miny, d_Maxy, d_Minz, d_Maxz);
      
      CudaErrorCheck(); 

      hipMemset(d_NoofNNlist, 0, 1024*1024);

      hipMemcpy(Minx, d_Minx, 6*sizeof(float), hipMemcpyDeviceToHost);
      Xdiv = (int)((Minx[1]-Minx[0])/DL+1);
      Ydiv = (int)((Minx[3]-Minx[2])/DL+1);
      Zdiv = (int)((Minx[5]-Minx[4])/DL+1);

      makeNNlist<<<No_of_C180s/512+1,512>>>( No_of_C180s, d_bounding_xyz, Minx[0], Minx[2], Minx[4],
                                             attraction_range, Xdiv, Ydiv, Zdiv, d_NoofNNlist, d_NNlist, DL);
      CudaErrorCheck();

      if (!growthDone && step > Time_steps+1){
          printf("Cell growth halted.\nProceeding with MD simulation without growth...\n");
          growthDone = true;
          
          if (useDifferentStiffnesses && !duringGrowth){
              printf("Now making some cells softer...\n");
              int softCellCounter = 0;
              if (fractionOfSofterCells > 0){
                  numberOfSofterCells = roundf(fractionOfSofterCells*No_of_C180s); 
              }

              printf("Will make %d cells softer\n", numberOfSofterCells); 
              
              if (chooseRandomCellIndices){
                  float rnd[1];
                  //int* chosenIndices = (int*)malloc(numberOfSofterCells, sizeof(int));
                  int chosenIndices[numberOfSofterCells]; 
                  
                  for (int i = 0; i < numberOfSofterCells; i++){
                      chosenIndices[i] = -1; 
                  }
                  
                  bool indexChosen = false;
                  int cellInd = -1;

                  printf("Make cells with indices "); 
                  
                  while (softCellCounter < numberOfSofterCells){
                      ranmar(rnd, 1);
                      cellInd = roundf(rnd[0] * No_of_C180s);

                      for (int i = 0; i < softCellCounter; i++){
                          if (chosenIndices[i] == cellInd){
                              indexChosen = true;
                              break;
                          }
                      }

                      if (!indexChosen){
                          chosenIndices[softCellCounter] = cellInd;
                          softCellCounter++;
                          printf("%d, ", cellInd); 
                      } else
                          indexChosen = false;
                      
                  }

                  //free(chosenIndices);

                  for (int i = 0; i < numberOfSofterCells; i++){
                      youngsModArray[chosenIndices[i]] = stiffness2; 
                  }
              }
              else {
                  // search for the oldest cells near the center of the system, and make them soft
                  hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
                  hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
                  hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);

                  float Rmax2 = getRmax2();
                  float R2, dx, dy, dz;
                  int cellInd = 0; 
                  calc_sys_CM();

                  float f = 1 - closenessToCenter;
              
                  printf("Made cells with indices "); 

                  while (softCellCounter < numberOfSofterCells && cellInd < No_of_C180s){
                      dx = CMx[cellInd] - sysCMx; 
                      dy = CMy[cellInd] - sysCMy; 
                      dz = CMz[cellInd] - sysCMz;

                      R2 = dx*dx + dy*dy + dz*dz;

                      if (R2 <= f*f*Rmax2){
                          printf("%d, ", cellInd); 
                          softCellCounter++; 
                          youngsModArray[cellInd] = stiffness2; 

                      }
                      cellInd++; 
                  }
              }
              
              hipMemcpy(d_Youngs_mod, youngsModArray, No_of_C180s*sizeof(float), hipMemcpyHostToDevice);
              printf("\b\b softer\n"); 
          }

      }

      if ( step%trajWriteInt == 0 )
      {
          //printf("   Writing trajectory to traj.xyz...\n");
          hipMemcpy(X, d_X, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(Y, d_Y, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(Z, d_Z, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);

          write_traj(step, trajfile);

          hipMemcpy(velListX, d_velListX, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(velListY, d_velListY, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          hipMemcpy(velListZ, d_velListZ, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
          
          write_vel(step, velFile); 
          // WriteToBinaryFile(X, Y, Z,
          //                   No_of_C180s, step, &bFA);
      }

#if defined(FORCE_DEBUG) || defined(PRINT_VOLUMES)
      volumes<<<No_of_C180s,192>>>(No_of_C180s, d_C180_56,
                                     d_XP, d_YP, d_ZP,
                                     d_CMx , d_CMy, d_CMz,
                                     d_volume, d_cell_div, divVol*100);
      
      hipMemcpy(volume, d_volume, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
      for (int i = 0; i < No_of_C180s; i++){
          printf ("Cell: %d, volume= %f\n", i, volume[i]);
      }
#endif



      myError = hipGetLastError();
      if ( hipSuccess != myError )
      {
          printf( "Error %d: %s!\n",myError,hipGetErrorString(myError) );return(-1);
      }
  }


  // Write postscript file
  //hipMemcpy(X, d_X, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
  //hipMemcpy(Y, d_Y, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
  //hipMemcpy(Z, d_Z, 192*No_of_C180s*sizeof(float),hipMemcpyDeviceToHost);
  //PSNET(No_of_C180s*270,Side_length,L1,X,Y,Z,CCI);

  printf("Xdiv = %d, Ydiv = %d, Zdiv = %d\n", Xdiv, Ydiv, Zdiv );

  FILE* MitIndFile;
  std::fstream MitIndFile2;
  std::string datFileName = inpFile; 
  
  if (overWriteMitInd == 0){
      
      MitIndFile = fopen(mitIndFileName, "a");
      //MitIndFile2.open(datFileName, "a"); 
  }
  else{
      MitIndFile = fopen(mitIndFileName, "w");
      //MitIndFile2.open(datFileName, "w"); 
  }
  if (MitIndFile == NULL)
  {
      printf("Failed to open mit-index.dat\n");
      exit(1);
  }


  for (int i = 0; i < (Time_steps/newCellCountInt) + 1; i++)
  {
      if ( dividingCells[i]!=0 && totalCells[i]!=0 ){
          fprintf(MitIndFile, "%f\n", (float)dividingCells[i]/totalCells[i]);
          // totalCells is number of non-dividing cells
          
      }
      else {
          fprintf(MitIndFile, "%f\n", 0.0);

      }

  }

  hipFree( (void *)d_bounding_xyz );
  hipFree( (void *)d_XP );
  hipFree( (void *)d_YP );
  hipFree( (void *)d_ZP );
  hipFree( (void *)d_X  );
  hipFree( (void *)d_Y  );
  hipFree( (void *)d_Z  );
  hipFree( (void *)d_XM );
  hipFree( (void *)d_YM );
  hipFree( (void *)d_ZM );
  hipFree( (void *)d_CMx );
  hipFree( (void *)d_CMy );
  hipFree( (void *)d_CMz );
  hipFree( (void *)d_ran2 );

  hipFree( (void *)d_C180_nn);
  hipFree( (void *)d_C180_sign);
  hipFree( (void *)d_cell_div);
  free(X); free(Y); free(Z);
  free(bounding_xyz);
  free(CMx); free(CMy); free(CMz);
  free(dividingCells); free(totalCells);
  free(NDIV);
  free(volume);
  free(Minx); free(Miny); free(Minz);
  free(Maxx); free(Maxy); free(Maxz);
  free(NoofNNlist);
  free(NNlist);
  free(ran2);
  free(num_new_cells_per_step);
  free(cell_div_inds);
  free(pressList);

  free(velListX); 
  free(velListY); 
  free(velListZ); 

  fclose(trajfile);
  fclose(MitIndFile);
  // CloseBinaryFile(&bFA);
  return(0);

}



int initialize_C180s(int Orig_No_of_C180s)
{
  int rank;
  int atom;
  float initx[181], inity[181], initz[181];
  FILE *infil;

  printf("      Initializing positions for %d fullerenes...\n", Orig_No_of_C180s);

  X = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  Y = (float *)calloc(192*MaxNoofC180s,sizeof(float));
  Z = (float *)calloc(192*MaxNoofC180s,sizeof(float));

  bounding_xyz = (float *)calloc(MaxNoofC180s,6*sizeof(float));

  CPUMemory += 3L*192L*MaxNoofC180s*sizeof(float);
  CPUMemory += MaxNoofC180s*6L*sizeof(float);

  infil = fopen("C180","r");
  if ( infil == NULL ) {printf("Unable to open file C180\n");return(-1);}
  for ( atom = 0 ; atom < 180 ; ++atom)
  {
      if ( fscanf(infil,"%f %f %f",&initx[atom], &inity[atom], &initz[atom]) != 3 )
      {
          printf("   Unable to read file C180 on line %d\n",atom+1);
          fclose(infil);
          return(-1);
      }
  }
  fclose(infil);

  ranmar(ran2,Orig_No_of_C180s);

  for ( rank = 0; rank < Orig_No_of_C180s; ++rank )
  {
      ey=rank%Side_length;
      ex=rank/Side_length;

      if (ranZOffset) // Branch prediction should kick in
          zOffset += ran2[rank]-0.5;

      for ( atom = 0 ; atom < 180 ; ++atom)
      {
          X[rank*192+atom] = initx[atom] + L1*ex + 0.5*L1;
          Y[rank*192+atom] = inity[atom] + L1*ey + 0.5*L1;
          Z[rank*192+atom] = initz[atom] + zOffset;
      }
  }

  return(0);
}


int generate_random(int no_of_ran1_vectors)
{
  // This function uses marsaglia random number generator
  // Defined in marsaglia.h
  int seed_ij, seed_kl ,ij,kl;

  ran2 = (float *)calloc(MaxNoofC180s+1,sizeof(float));
  CPUMemory += (MaxNoofC180s+1L)*sizeof(float);

  time_t current_time;
  time(&current_time);
  seed_ij = (int)current_time;
  localtime(&current_time);
  seed_kl = (int)current_time;
  ij = seed_ij%31328;
  kl = seed_kl%30081;
  rmarin(ij,kl);

  printf("RNG seeds: %d, %d\n", ij, kl); 
  return(0);
}



int read_fullerene_nn(void)
{
  int i,end;
  int N1, N2, N3, N4, N5, N6, Sign;
  FILE *infil;

  printf("   Reading C180NN ..\n");

  infil = fopen("C180NN","r");
  if ( infil == NULL ) {printf("Unable to open file C180NN\n");return(-1);}

  end = 180;
  for ( i = 0; i < 180 ; ++i )
  {
      if ( fscanf(infil,"%d,%d,%d,%d", &N1, &N2, &N3, &Sign) != 4 ) {end = i; break;}
      C180_nn[0 + i] = N1-1;
      C180_nn[192+i] = N2-1;
      C180_nn[384+i] = N3-1;
      C180_sign[i] = Sign;
  }
  fclose(infil);

  if ( end < 180 ) {printf("Error: Unable to read line %d in file C180NN\n",end);return(-1);}

  printf("   Reading C180C ..\n");

  infil = fopen("C180C","r");
  if ( infil == NULL ) {printf("Unable to open file C180C\n");return(-1);}

  end = 270;
  for ( i = 0; i < 270 ; ++i )
  {
      if ( fscanf(infil,"%d,%d", &N1, &N2) != 2 ) {end = i; break;}
      CCI[0][i] = N1-1;
      CCI[1][i] = N2-1;
  }
  fclose(infil);

  if ( end < 270 ) {printf("Error: Unable to read line %d in file C180C\n",end);return(-1);}

  printf("      read nearest neighbour ids for atoms in C180\n");

  printf("   Reading C180 pentagons, hexagons ..\n");

  infil = fopen("C180_pentahexa","r");
  if ( infil == NULL ) {printf("Unable to open file C180_pentahexa\n");return(-1);}

  end = 12;
  for ( i = 0; i < 12 ; ++i )
  {
      if ( fscanf(infil,"%d %d %d %d %d", &N1, &N2, &N3, &N4, &N5) != 5 ) {end = i; break;}
      C180_56[i*7+0] = N1;
      C180_56[i*7+1] = N2;
      C180_56[i*7+2] = N3;
      C180_56[i*7+3] = N4;
      C180_56[i*7+4] = N5;
      C180_56[i*7+5] = N1;
      C180_56[i*7+6] = N1;
  }
  if ( end != 12 ) {printf("Error: Unable to read line %d in file C180_pentahexa\n",end);return(-1);}
  end = 80;
  for ( i = 0; i < 80 ; ++i )
  {
      if ( fscanf(infil,"%d %d %d %d %d %d", &N1, &N2, &N3, &N4, &N5, &N6) != 6 ) {end = i; break;}
      C180_56[84+i*7+0] = N1;
      C180_56[84+i*7+1] = N2;
      C180_56[84+i*7+2] = N3;
      C180_56[84+i*7+3] = N4;
      C180_56[84+i*7+4] = N5;
      C180_56[84+i*7+5] = N6;
      C180_56[84+i*7+6] = N1;
  }
  if ( end != 80 ) {printf("Error: Unable to read line %d in file C180_pentahexa\n",end);return(-1);}

  fclose(infil);

  return(0);
}


int read_json_params(const char* inpFile){
    // Function to parse a json input file using the jsoncpp library

    // variable to hold the root of the json input
    Json::Value inpRoot;
    Json::Reader inpReader;

    std::ifstream inpStream(inpFile);
    std::string inpString((std::istreambuf_iterator<char>(inpStream)),
                          std::istreambuf_iterator<char>());

    bool parsingSuccess = inpReader.parse(inpString, inpRoot);
    if (!parsingSuccess){
        printf("Failed to parse %s\n", inpFile);
        // There must be a way to keep from converting from string to char*
        // Maybe by making inpString a char*
        printf("%s", inpReader.getFormattedErrorMessages().c_str());
        return -1;
    }
    else
        printf("%s parsed successfully\n", inpFile);

    // begin detailed parameter extraction

    Json::Value coreParams = inpRoot.get("core", Json::nullValue);

    // load core simulation parameters
    if (coreParams == Json::nullValue){
        printf("ERROR: Cannot load core simulation parameters\nExiting");
        return -1;
    }
    else {
        MaxNoofC180s = coreParams["MaxNoofC180s"].asInt(); 
        mass = coreParams["particle_mass"].asFloat();
        repulsion_range = coreParams["repulsion_range"].asFloat();
        attraction_range = coreParams["attraction_range"].asFloat();
        repulsion_strength = coreParams["repulsion_strength"].asFloat();
        attraction_strength = coreParams["attraction_strength"].asFloat();
        stiffness1 = coreParams["Youngs_mod"].asFloat();
        viscotic_damping = coreParams["viscotic_damping"].asFloat();
        internal_damping = coreParams["internal_damping"].asFloat();
        divVol = coreParams["division_Vol"].asFloat();
        ranZOffset = coreParams["random_z_offset?"].asInt();
        zOffset = coreParams["z_offset"].asFloat();
        Time_steps = coreParams["div_time_steps"].asFloat();
        delta_t = coreParams["time_interval"].asFloat();
        Restart = coreParams["Restart"].asInt();
        trajWriteInt = coreParams["trajWriteInt"].asInt();
        equiStepCount = coreParams["non_div_time_steps"].asInt();

        std::strcpy (trajFileName, coreParams["trajFileName"].asString().c_str());

        maxPressure = coreParams["maxPressure"].asFloat();
        minPressure = coreParams["minPressure"].asFloat();
        gamma_visc = coreParams["gamma_visc"].asFloat();
        rMax = coreParams["growth_rate"].asFloat();     

    }

    Json::Value countParams = inpRoot.get("counting", Json::nullValue);
    if (countParams == Json::nullValue){
        // countCells = FALSE;
        printf("ERROR: Cannot load counting parameters\nExiting");
        return -1;
    }
    else {
        // countCells = countParams["countcells"].asBool();
        std::strcpy(mitIndFileName, countParams["mit-index_file_name"].asString().c_str()); 
        countOnlyInternal = countParams["count_only_internal_cells?"].asBool();
        radFrac = countParams["radius_cutoff"].asFloat();
        overWriteMitInd = countParams["overwrite_mit_ind_file?"].asBool();
        newCellCountInt = countParams["cell_count_int"].asInt();
    }

    Json::Value popParams = inpRoot.get("population", Json::nullValue);
    if (popParams == Json::nullValue){
        printf("ERROR: Cannot load population parameters\nExiting");
        return -1;
    }
    else{
        doPopModel = popParams["doPopModel"].asInt();
        totalFood = popParams["totalFood"].asFloat();
        cellFoodCons = popParams["regular_consumption"].asFloat();
        cellFoodConsDiv = popParams["division_consumption"].asFloat();
        cellFoodRel = popParams["death_release_food"].asFloat();
        cellLifeTime = popParams["cellLifeTime"].asInt();
        maxPop = popParams["max_pop"].asFloat(); 
    }

    Json::Value wallParams = inpRoot.get("walls", Json::nullValue);

    if (wallParams == Json::nullValue){
        printf("ERROR: Cannot load wall parameters\nExiting");
        return -1;
    }
    else{
        useWalls = wallParams["useWalls"].asInt();
        std::strcpy(perpAxis, wallParams["perpAxis"].asString().c_str());
        dAxis = wallParams["dAxis"].asFloat();
        wallLen = wallParams["wallLen"].asFloat();
        wallWidth = wallParams["wallWidth"].asFloat();
        threshDist = wallParams["threshDist"].asFloat();
    }

    Json::Value divParams = inpRoot.get("divParams", Json::nullValue);
    
    if (divParams == Json::nullValue){
        printf("ERROR: Cannot load division parameters\n");
        return -1;
    } else{
        useDivPlaneBasis = divParams["useDivPlaneBasis"].asInt();
        divPlaneBasis[0] = divParams["divPlaneBasisX"].asFloat();
        divPlaneBasis[1] = divParams["divPlaneBasisY"].asFloat();
        divPlaneBasis[2] = divParams["divPlaneBasisZ"].asFloat();
    }

    Json::Value stiffnessParams = inpRoot.get("stiffnessParams", Json::nullValue);

    if (stiffnessParams == Json::nullValue){
        printf("ERROR: Cannot load stiffness parameters\n");
        return -1;
    } else {
        useDifferentStiffnesses = stiffnessParams["useDifferentStiffnesses"].asBool();
        stiffness2 = stiffnessParams["softYoungsMod"].asFloat();
        numberOfSofterCells = stiffnessParams["numberOfSofterCells"].asInt();
        duringGrowth = stiffnessParams["duringGrowth"].asBool(); 
        closenessToCenter = stiffnessParams["closenessToCenter"].asFloat();
        startAtPop = stiffnessParams["startAtPop"].asInt();
        fractionOfSofterCells = stiffnessParams["fractionOfSofterCells"].asFloat();
        chooseRandomCellIndices = stiffnessParams["chooseRandomCellIndices"].asBool();
        daughtSameStiffness = stiffnessParams["daughtSameStiffness"].asBool(); 
    }

    Json::Value boxParams = inpRoot.get("boxParams", Json::nullValue);

    if (boxParams == Json::nullValue){
        printf("ERROR: Cannot load box parameters\n");
        return -1;
    } else{
        useRigidSimulationBox = boxParams["useRigidSimulationBox"].asBool();
    }
    
    
    if (ranZOffset == 0)
        zOffset = 0.0;


    printf("      mass                = %f\n",mass);
    printf("      spring equilibrium  = %f\n",R0);
    printf("      repulsion range     = %f\n",repulsion_range);
    printf("      attraction range    = %f\n",attraction_range);
    printf("      repulsion strength  = %f\n",repulsion_strength);
    printf("      attraction strength = %f\n",attraction_strength);
    printf("      Youngs modulus      = %f\n",stiffness1);
    printf("      viscotic damping    = %f\n",viscotic_damping);
    printf("      internal damping    = %f\n",internal_damping);
    printf("      division volume     = %f\n",divVol);
    printf("      ran_z_offset?       = %d\n", ranZOffset);
    printf("      z_offset            = %f\n", zOffset);
    printf("      Time steps          = %d\n",Time_steps);
    printf("      delta t             = %f\n",delta_t);
    printf("      Restart             = %d\n",Restart);
    printf("      trajWriteInterval   = %d\n",trajWriteInt);
    printf("      countOnlyInternal   = %d\n", countOnlyInternal);
    printf("      radFrac             = %f\n", radFrac);
    printf("      newCellCountInt     = %d\n", newCellCountInt);
    printf("      equiStepCount       = %d\n", equiStepCount);
    printf("      trajFileName        = %s\n", trajFileName);
    printf("      doPopModel          = %d\n", doPopModel);
    printf("      totalFood           = %f\n", totalFood);
    printf("      cellFoodCons        = %f\n", cellFoodCons);
    printf("      cellFoodConsDiv     = %f\n", cellFoodConsDiv);
    printf("      cellFoodRel         = %f\n", cellFoodRel);
    printf("      useWalls            = %d\n", useWalls);
    printf("      perpAxis            = %s\n", perpAxis);
    printf("      dAxis               = %f\n", dAxis);
    printf("      wallLen             = %f\n", wallLen);
    printf("      wallWidth           = %f\n", wallWidth);
    printf("      thresDist           = %f\n", threshDist);
    printf("      maxPressure         = %f\n", maxPressure);
    printf("      minPressure         = %f\n", minPressure);
    printf("      gamma_visc          = %f\n", gamma_visc);
    printf("      useDivPlanebasis    = %d\n", useDivPlaneBasis);
    printf("      divPlaneBasisX      = %f\n", divPlaneBasis[0]);
    printf("      divPlaneBasisY      = %f\n", divPlaneBasis[1]);
    printf("      divPlaneBasisZ      = %f\n", divPlaneBasis[2]);
    printf("      useDifferentStiffnesses = %d\n", useDifferentStiffnesses);
    printf("      softYoungsMod       = %f\n", softYoungsMod);
    printf("      numberOfsofterCells = %d\n", numberOfSofterCells);
    printf("      duringGrowth        = %d\n", duringGrowth);
    printf("      closenesstoCenter   = %f\n", closenessToCenter);
    printf("      startAtPop          = %d\n", startAtPop);
    printf("      fractionOfSofterCells   = %f\n", fractionOfSofterCells);
    printf("      chooseRandomCellIndices = %d\n", chooseRandomCellIndices);
    printf("      daughtSameStiffness = %d\n", daughtSameStiffness);
    printf("      useRigidSimulationBox = %d\n", useRigidSimulationBox);
    
    



    if ( radFrac < 0.4 || radFrac > 0.8 || radFrac < 0 ){
        printf("radFrac not in [0.4, 0.8] setting to 1.\n");
        countOnlyInternal = 0;
    }

    if (trajWriteInt == 0){
        trajWriteInt = 1;
    }

    if (newCellCountInt == 0){
        newCellCountInt = 1;
    }

    if ( trajWriteInt > Time_steps + equiStepCount){
        printf ("Trajectory write interval is too large\n");
        return -1;
    }

    if (Time_steps%trajWriteInt != 0){
        printf ("Invalid trajectory write interval. Time steps must be divisible by it. \n");
        return -1;
    }

    if (newCellCountInt > Time_steps){
        printf("New cell counting interval is too large. \n");
        return -1;
    }

    if (equiStepCount <= 0){
        equiStepCount = 0;
    }

    if (doPopModel != 1){ // This ensures that Pop modelling is only done if this
        // var is only 1
        doPopModel = 0;
    }

    if (maxPressure < 0){
        printf("Invalid maximum pressure value of %f\n", maxPressure);
        printf("Disabling population modelling...");
        doPopModel = 0;
    }


    /*

    // The if statement below is not a very good one
    // think about rewriting.
    if (totalFood < 0.0
    || No_of_threads*100 < totalFood
    || cellFoodCons < 0.0
    || cellFoodCons*No_of_threads*10 < totalFood
    || cellFoodConsDiv < 0.0
    || cellFoodConsDiv*No_of_threads*10 < totalFood
    ){
    doPopModel = 0;
    printf("Food parameters invalid. Skipping population modelling.\n");
    }
    */

    if ( !(closenessToCenter >=0 && closenessToCenter <= 1) ){
        printf("ERROR: closenessToCenter is not in [0, 1]\n");
        printf("ERROR: invalid input parameter\n");
        return -1;
    }

    if (useWalls && useRigidSimulationBox){
        printf("ERROR: Cannot use infinite XY walls and rigid simulation box simultaneously.\n");
        printf("ERROR: Only use on or the other.\n");
        return -1;
    }

    if (fractionOfSofterCells > 1.0){
        printf("ERROR: Softer cell fraction is > 1\n");
        return -1;
    }
        

    return 0;
}


int read_global_params(void)
{
  int error;
  FILE *infil;

  printf("   Reading inp.dat ..\n");

  infil = fopen("inp.dat","r");
  if ( infil == NULL ) {printf("Error: Unable to open file inp.dat\n");return(-1);}

  error = 0;


  if ( fscanf(infil,"%f",&mass)                != 1 ) {error =  1 ;}
  if ( fscanf(infil,"%f",&repulsion_range)     != 1 ) {error =  2 ;}
  if ( fscanf(infil,"%f",&attraction_range)    != 1 ) {error =  3 ;}
  if ( fscanf(infil,"%f",&repulsion_strength)  != 1 ) {error =  4 ;}
  if ( fscanf(infil,"%f",&attraction_strength) != 1 ) {error =  5 ;}
//  if ( fscanf(infil,"%f",&Youngs_mod)          != 1 ) {error =  6 ;}
  if ( fscanf(infil,"%f",&viscotic_damping)    != 1 ) {error =  7 ;}
  if ( fscanf(infil,"%f",&internal_damping)    != 1 ) {error =  8 ;}
  if ( fscanf(infil,"%f",&divVol)              != 1 ) {error =  9 ;}
  if ( fscanf(infil,"%d",&Time_steps)          != 1 ) {error = 10 ;}
  if ( fscanf(infil,"%f",&delta_t)             != 1 ) {error = 11 ;}
  if ( fscanf(infil,"%d",&Restart)             != 1 ) {error = 12 ;}
  if ( fscanf(infil,"%d",&trajWriteInt)        != 1 ) {error = 13 ;}
  if ( fscanf(infil,"%d",&countOnlyInternal)   != 1 ) {error = 14 ;}
  if ( fscanf(infil,"%f",&radFrac)             != 1 ) {error = 15 ;}
  if ( fscanf(infil,"%d",&overWriteMitInd)     != 1 ) {error = 16 ;}
  if ( fscanf(infil,"%d",&newCellCountInt)     != 1 ) {error = 17 ;}
  if ( fscanf(infil,"%d",&equiStepCount)       != 1 ) {error = 18 ;}
  if ( fscanf(infil,"%s",trajFileName)         != 1 ) {error = 19 ;}
  if ( fscanf(infil,"%d",&doPopModel)          != 1 ) {error = 20 ;}
  if ( fscanf(infil,"%f",&totalFood)           != 1 ) {error = 21 ;}
  if ( fscanf(infil,"%f",&cellFoodCons)        != 1 ) {error = 22 ;}
  if ( fscanf(infil,"%f",&cellFoodConsDiv)     != 1 ) {error = 23 ;}
  if ( fscanf(infil,"%f",&cellFoodRel)         != 1 ) {error = 24 ;}
  if ( fscanf(infil,"%d",&haylimit)            != 1 ) {error = 25 ;}
  if ( fscanf(infil,"%d",&cellLifeTime)        != 1 ) {error = 26 ;}
  if ( fscanf(infil,"%f",&maxPressure)         != 1 ) {error = 27 ;}
  if ( fscanf(infil,"%d",&useWalls)            != 1 ) {error = 28 ;}
  if ( fscanf(infil,"%s",perpAxis)             != 1 ) {error = 29 ;}
  if ( fscanf(infil,"%f",&dAxis)               != 1 ) {error = 30 ;}
  if ( fscanf(infil,"%f",&wallLen)             != 1 ) {error = 31 ;}
  if ( fscanf(infil,"%f",&wallWidth)           != 1 ) {error = 32 ;}
  if ( fscanf(infil,"%f",&threshDist)          != 1 ) {error = 33 ;}





  fclose(infil);

  if ( error != 0 ){
      printf("   Error reading line %d from file inp.dat\n",error);
      return(-1);
  }

  if ( radFrac < 0.4 || radFrac > 0.8 || radFrac < 0 ){
      printf("radFrac not in [0.4, 0.8] setting to 1.\n");
      countOnlyInternal = 0;
  }

  if (trajWriteInt == 0){
      trajWriteInt = 1;
  }

  if (newCellCountInt == 0){
      newCellCountInt = 1;
  }

  if ( trajWriteInt > Time_steps){
      printf ("Trajectory write interval is too large\n");
      return -1;
  }

  if (Time_steps%trajWriteInt != 0){
      printf ("Invalid trajectory write interval. Time steps must be divisible by it. \n");
      return -1;
  }

  if (newCellCountInt > Time_steps){
      printf("New cell counting interval is too large. \n");
      return -1;
  }

  if (equiStepCount <= 0){
    equiStepCount = 0;
  }

  if (doPopModel != 1){ // This ensures that Pop modelling is only done if this
                        // var is only 1
      doPopModel = 0;
  }

  if (maxPressure < 0){
      printf("Invalid maximum pressure value of %f\n", maxPressure);
      printf("Disabling population modelling...");
      doPopModel = 0;
  }


  /*

  // The if statement below is not a very good one
  // think about rewriting.
  if (totalFood < 0.0
      || No_of_threads*100 < totalFood
      || cellFoodCons < 0.0
      || cellFoodCons*No_of_threads*10 < totalFood
      || cellFoodConsDiv < 0.0
      || cellFoodConsDiv*No_of_threads*10 < totalFood
       ){
      doPopModel = 0;
      printf("Food parameters invalid. Skipping population modelling.\n");
  }
  */


  printf("      mass                = %f\n",mass);
  printf("      spring equilibrium  = %f\n",R0);
  printf("      repulsion range     = %f\n",repulsion_range);
  printf("      attraction range    = %f\n",attraction_range);
  printf("      repulsion strength  = %f\n",repulsion_strength);
  printf("      attraction strength = %f\n",attraction_strength);
//  printf("      Youngs modulus      = %f\n",Youngs_mod);
  printf("      viscotic damping    = %f\n",viscotic_damping);
  printf("      internal damping    = %f\n",internal_damping);
  printf("      division volume     = %f\n",divVol);
  printf("      Time steps          = %d\n",Time_steps);
  printf("      delta t             = %f\n",delta_t);
  printf("      Restart             = %d\n",Restart);
  printf("      trajWriteInterval   = %d\n",trajWriteInt);
  printf("      countOnlyInternal   = %d\n", countOnlyInternal);
  printf("      radFrac             = %f\n", radFrac);
  printf("      newCellCountInt     = %d\n", newCellCountInt);
  printf("      equiStepCount       = %d\n", equiStepCount);
  printf("      trajFileName        = %s\n", trajFileName);
  printf("      doPopModel          = %d\n", doPopModel);
  printf("      totalFood           = %f\n", totalFood);
  printf("      cellFoodCons        = %f\n", cellFoodCons);
  printf("      cellFoodConsDiv     = %f\n", cellFoodConsDiv);
  printf("      cellFoodRel         = %f\n", cellFoodRel);
  printf("      useWalls            = %d\n", useWalls);
  printf("      perpAxis            = %s\n", perpAxis);
  printf("      dAxis               = %f\n", dAxis);
  printf("      wallLen             = %f\n", wallLen);
  printf("      wallWidth           = %f\n", wallWidth);
  printf("      thresDist           = %f\n", threshDist);


  return(0);
}




//C *****************************************************************



__global__ void propagate( int No_of_C180s, int d_C180_nn[], int d_C180_sign[],
                           float d_XP[], float d_YP[], float d_ZP[],
                           float d_X[],  float d_Y[],  float d_Z[],
                           float d_XM[], float d_YM[], float d_ZM[],
                           float *d_CMx, float *d_CMy, float *d_CMz,
                           float R0, float* d_pressList, float* d_Youngs_mod , float bondingYoungsMod, 
                           float internal_damping, float delta_t,
                           float d_bounding_xyz[],
                           float attraction_strength, float attraction_range,
                           float repulsion_strength, float repulsion_range,
                           float viscotic_damping, float mass,
                           float Minx, float Miny,  float Minz, int Xdiv, int Ydiv, int Zdiv,
                           int *d_NoofNNlist, int *d_NNlist, float DL, float gamma_visc,
                           float wall1, float wall2,
                           float threshDist, bool useWalls, 
                           float* d_velListX, float* d_velListY, float* d_velListZ,
                           bool useRigidSimulationBox, float boxLength, float* d_boxMin)
{
#ifdef FORCE_DEBUG
        __shared__ float FX_sum;
        __shared__ float FY_sum;
        __shared__ float FZ_sum;
        if (threadIdx.x == 0){
            FX_sum = 0;
            FY_sum = 0;
            FZ_sum = 0;
        }

        __syncthreads();

#endif

    int rank, atom, nn_rank, nn_atom;
    int N1, N2, N3;
    int NooflocalNN;
    int localNNs[10];
    float deltaX, deltaY, deltaZ;
    float A1, A2, A3;
    float B1, B2, B3;
    float TX, TY, TZ;
    float NORM, R;
    float NX, NY, NZ;
    //float setPress;

    rank = blockIdx.x;
    atom = threadIdx.x;
    float Pressure = d_pressList[rank]; 
    int cellOffset = rank*192;
    int atomInd = cellOffset + atom;
    float Youngs_mod;
    
    if ( rank < No_of_C180s && atom < 180 )
    {
        if (isnan(d_X[rank*192+atom]) ||
            isnan(d_Y[rank*192+atom]) || 
            isnan(d_Z[rank*192+atom])){
            printf("OH SHIT: we have a nan\n");
            printf("Particle index: %d\n", atom);
            printf("Crash now :(\n"); 
            asm("trap;"); 
        }

        Youngs_mod = d_Youngs_mod[rank];
        
        N1 = d_C180_nn[  0+atom];
        N2 = d_C180_nn[192+atom];
        N3 = d_C180_nn[384+atom];

        A1 = d_X[rank*192+N2]-d_X[rank*192+N1];
        A2 = d_Y[rank*192+N2]-d_Y[rank*192+N1];
        A3 = d_Z[rank*192+N2]-d_Z[rank*192+N1];

        B1 = d_X[rank*192+N3]-d_X[rank*192+N1];
        B2 = d_Y[rank*192+N3]-d_Y[rank*192+N1];
        B3 = d_Z[rank*192+N3]-d_Z[rank*192+N1];

        TX = A2*B3-A3*B2;
        TY = A3*B1-A1*B3;
        TZ = A1*B2-A2*B1;

        NORM = sqrt(TX*TX+TY*TY+TZ*TZ);

        NX = d_C180_sign[atom]*TX/NORM;
        NY = d_C180_sign[atom]*TY/NORM;
        NZ = d_C180_sign[atom]*TZ/NORM;

        float X = d_X[rank*192+atom];
        float Y = d_Y[rank*192+atom];
        float Z = d_Z[rank*192+atom];

        float FX = 0.0f;
        float FY = 0.0f;
        float FZ = 0.0f;

        int nnAtomInd;
        
        
        float velX = d_velListX[atomInd];
        float velY = d_velListY[atomInd];
        float velZ = d_velListZ[atomInd];


        //  Spring Force calculation within cell
        //  go through three nearest neighbors

        float damp_const = internal_damping/delta_t;

        for ( int i = 0; i < 3 ; ++i ) // Better to open this loop
        {
            N1 = d_C180_nn[i*192+atom];

            deltaX = d_X[rank*192+N1]-d_X[rank*192+atom];
            deltaY = d_Y[rank*192+N1]-d_Y[rank*192+atom];
            deltaZ = d_Z[rank*192+N1]-d_Z[rank*192+atom];

            R  = sqrt(deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ);

            // spring forces
            FX += +Youngs_mod*(R-R0)/R0*deltaX/R;
            FY += +Youngs_mod*(R-R0)/R0*deltaY/R;
            FZ += +Youngs_mod*(R-R0)/R0*deltaZ/R;


            // pressure forces
            FX += Pressure*NX;
            FY += Pressure*NY;
            FZ += Pressure*NZ;

            // internal damping
            // FX += -damp_const*(-deltaX-(d_XM[rank*192+atom]-d_XM[rank*192+N1]));
            // FY += -damp_const*(-deltaY-(d_YM[rank*192+atom]-d_YM[rank*192+N1]));
            // FZ += -damp_const*(-deltaZ-(d_ZM[rank*192+atom]-d_ZM[rank*192+N1]));

            FX += -internal_damping*(d_velListX[atomInd] - d_velListX[rank*192+N1]);
            FY += -internal_damping*(d_velListY[atomInd] - d_velListY[rank*192+N1]);
            FZ += -internal_damping*(d_velListZ[atomInd] - d_velListZ[rank*192+N1]);

        }

#ifdef FORCE_DEBUG

        atomicAdd(&FX_sum, FX);
        __syncthreads();
        atomicAdd(&FY_sum, FY);
        __syncthreads();
        atomicAdd(&FZ_sum, FZ);
        __syncthreads();
        if (threadIdx.x == 0){
            printf("Spring, pressure, internal\n");
            printf("Fx = %f, Fy = %f, Fz = %f\n", FX_sum, FY_sum, FZ_sum);
        }

#endif


        NooflocalNN = 0;

        int startx = (int)((X -Minx)/DL);
        if ( startx < 0 ) startx = 0;
        if ( startx >= Xdiv ) startx = Xdiv-1;

        int starty = (int)((Y - Miny)/DL);
        if ( starty < 0 ) starty = 0;
        if ( starty >= Ydiv ) starty = Ydiv-1;

        int startz = (int)((Z - Minz)/DL);
        if ( startz < 0 ) startz = 0;
        if ( startz >= Zdiv ) startz = Zdiv-1;

        int index = startz*Xdiv*Ydiv + starty*Xdiv + startx;

        // interfullerene attraction and repulsion
        for ( int nn_rank1 = 1 ; nn_rank1 <= d_NoofNNlist[index] ; ++nn_rank1 )
        {
            nn_rank = d_NNlist[32*index+nn_rank1-1];
            if ( nn_rank == rank ) continue;

            deltaX  = (X-d_bounding_xyz[nn_rank*6+1]>0.0f)*(X-d_bounding_xyz[nn_rank*6+1]);
            deltaX += (d_bounding_xyz[nn_rank*6+0]-X>0.0f)*(d_bounding_xyz[nn_rank*6+0]-X);

            deltaY  = (Y-d_bounding_xyz[nn_rank*6+3]>0.0f)*(Y-d_bounding_xyz[nn_rank*6+3]);
            deltaY += (d_bounding_xyz[nn_rank*6+2]-Y>0.0f)*(d_bounding_xyz[nn_rank*6+2]-Y);

            deltaZ  = (Z-d_bounding_xyz[nn_rank*6+5]>0.0f)*(Z-d_bounding_xyz[nn_rank*6+5]);
            deltaZ += (d_bounding_xyz[nn_rank*6+4]-Z>0.0f)*(d_bounding_xyz[nn_rank*6+4]-Z);

            if ( deltaX*deltaX + deltaY*deltaY + deltaZ*deltaZ > attraction_range*attraction_range )
                continue;

            ++NooflocalNN;

            if ( NooflocalNN > 10 ){
                printf("Recoverable error: NooflocalNN = %d, should be < 8\n",NooflocalNN);
                continue;
            }

            localNNs[NooflocalNN-1] = nn_rank;
        }

        for ( int i = 0; i < NooflocalNN; ++i )
        {
            nn_rank =localNNs[i];
            nnAtomInd = nn_rank*192;

            for ( nn_atom = 0; nn_atom < 180 ; ++nn_atom )
            {
                nnAtomInd += nn_atom;

                deltaX = d_X[rank*192+atom]-d_X[nn_rank*192+nn_atom];
                deltaY = d_Y[rank*192+atom]-d_Y[nn_rank*192+nn_atom];
                deltaZ = d_Z[rank*192+atom]-d_Z[nn_rank*192+nn_atom];

                R = deltaX*deltaX+deltaY*deltaY+deltaZ*deltaZ;

                if ( R >= attraction_range*attraction_range )
                    continue;

                R = sqrt(R);

                if ( R < attraction_range )
                {
                    FX += -attraction_strength*Youngs_mod*(attraction_range-R)/R*deltaX;
                    FY += -attraction_strength*Youngs_mod*(attraction_range-R)/R*deltaY;
                    FZ += -attraction_strength*Youngs_mod*(attraction_range-R)/R*deltaZ;

                    // hinder rearrangements

                    // First calculate relative velocity
                    float v_ijx = d_velListX[atomInd] - d_velListX[nn_rank*192+nn_atom];
                    float v_ijy = d_velListY[atomInd] - d_velListY[nn_rank*192+nn_atom];
                    float v_ijz = d_velListZ[atomInd] - d_velListZ[nn_rank*192+nn_atom];

                    // Dot product between relative and normal to surface
                    float vijDotn = v_ijx*NX + v_ijy*NY + v_ijz*NZ; 

                    // Tangential component
                    float vTauX = v_ijx - vijDotn*NX;
                    float vTauY = v_ijy - vijDotn*NY;
                    float vTauZ = v_ijz - vijDotn*NZ; 

                    FX -= viscotic_damping*vTauX;
                    FY -= viscotic_damping*vTauY;
                    FZ -= viscotic_damping*vTauZ;
                }
                if ( R < repulsion_range )
                {
                    if (R < (repulsion_range-0.01)) R = repulsion_range-0.01; 
                    FX += +repulsion_strength*Youngs_mod*(repulsion_range-R)/R*deltaX;
                    FY += +repulsion_strength*Youngs_mod*(repulsion_range-R)/R*deltaY;
                    FZ += +repulsion_strength*Youngs_mod*(repulsion_range-R)/R*deltaZ;

                    // if ( deltaX*(d_CMx[rank]-d_CMx[nn_rank])  +
                    //      deltaY*(d_CMy[rank]-d_CMy[nn_rank])  +
                    //      deltaZ*(d_CMz[rank]-d_CMz[nn_rank]) < 0.0f )
                    // {
                    //     //printf("fullerene %d inside %d?\n",rank, nn_rank);
                    // }
                }

            }

        }

#ifdef FORCE_DEBUG

        if (threadIdx.x == 0){
            FX_sum = 0;
            FY_sum = 0;
            FZ_sum = 0;
        }
        __syncthreads();

        atomicAdd(&FX_sum, FX);
        __syncthreads();
        atomicAdd(&FY_sum, FY);
        __syncthreads();
        atomicAdd(&FZ_sum, FZ);
        __syncthreads();
        if (threadIdx.x == 0){
            printf("neighbours\n");
            printf("Fx = %f, Fy = %f, Fz = %f\n", FX_sum, FY_sum, FZ_sum);
        }



#endif

        // add friction

        FX += -1 * gamma_visc * velX;
        FY += -1 * gamma_visc * velY;
        FZ += -1 * gamma_visc * velZ;

        if (useWalls == 1){
            // Wall repulsion

            float wallDist = d_Z[rank*192+atom] - wall1;
            
            if (abs(wallDist) <= threshDist){
            //repulse
                if (wallDist * FZ < 0) FZ = -FZ;
            }

            wallDist = d_Z[rank*192+atom] - wall2;
            if (abs(wallDist) <= threshDist){
                
                if (wallDist * FZ < 0) FZ = -FZ;

            }

            __syncthreads();
        }

        // add forces from simulation box if needed:

        if (useRigidSimulationBox){
            float gap1, gap2;
            
            // X
            
            gap1 = d_X[atomInd] - d_boxMin[0];
            gap2 = d_boxMin[0] + boxLength - d_X[atomInd];
            
            if (abs(gap1) < threshDist && gap1*FX < 0) FX = -FX;
            if (abs(gap2) < threshDist && gap2*FX < 0) FX = -FX;

            // Y

            gap1 = d_Y[atomInd] - d_boxMin[1];
            gap2 = d_boxMin[1] + boxLength - d_Y[atomInd];

            if (abs(gap1) < threshDist && gap1*FY < 0) FY = -FY;
            if (abs(gap2) < threshDist && gap2*FY < 0) FY = -FY;

            // Z
            gap1 = d_Z[atomInd] - d_boxMin[2];
            gap2 = d_boxMin[2] + boxLength - d_Z[atomInd];

            if (abs(gap1) < threshDist && gap1*FZ < 0) FZ = -FZ;
            if (abs(gap2) < threshDist && gap2*FZ < 0) FZ = -FZ;
        }

        // time propagation
        
        d_XP[rank*192+atom] =
            1.0/(1.0+delta_t/(2*mass))*
            ((delta_t*delta_t/mass)*FX+2*d_X[rank*192+atom]+(delta_t/(2*mass)-1.0)*d_XM[rank*192+atom]);
        d_YP[rank*192+atom] =
            1.0/(1.0+delta_t/(2*mass))*
            ((delta_t*delta_t/mass)*FY+2*d_Y[rank*192+atom]+(delta_t/(2*mass)-1.0)*d_YM[rank*192+atom]);
        d_ZP[rank*192+atom] =
            1.0/(1.0+delta_t/(2*mass))*
            ((delta_t*delta_t/mass)*FZ+2*d_Z[rank*192+atom]+(delta_t/(2*mass)-1.0)*d_ZM[rank*192+atom]);

        d_velListX[rank*192+atom] = (d_XP[atomInd] - d_XM[atomInd])/(2*delta_t); 
        d_velListY[rank*192+atom] = (d_YP[atomInd] - d_YM[atomInd])/(2*delta_t); 
        d_velListZ[rank*192+atom] = (d_ZP[atomInd] - d_ZM[atomInd])/(2*delta_t);

    }
}

void write_traj(int t_step, FILE* trajfile)
{

  fprintf(trajfile, "%d\n", No_of_C180s * 192);
  fprintf(trajfile, "Step: %d\n", t_step);
  
  if (useDifferentStiffnesses){
      
        for (int p = 0; p < No_of_C180s*192; p++)
        {
            int cellInd = p/192; 
            if (youngsModArray[cellInd] == stiffness1)
                fprintf(trajfile, "%.7f,  %.7f,  %.7f,  H\n", X[p], Y[p], Z[p]);
            else if(youngsModArray[cellInd] == stiffness2)
                fprintf(trajfile, "%.7f,  %.7f,  %.7f,  C\n", X[p], Y[p], Z[p]);

        }
        
  } else {
      
      for (int p = 0; p < No_of_C180s*192; p++)
      {
          fprintf(trajfile, "%.7f,  %.7f,  %.7f\n", X[p], Y[p], Z[p]);
      }
      
  }
}

void write_vel(int t_step, FILE* velFile){
    fprintf(velFile, "%d\n", No_of_C180s * 192);
    fprintf(velFile, "Step: %d\n", t_step);
    for (int p = 0; p < No_of_C180s*192; p++)
    {
        fprintf(velFile, "%.7f,  %.7f,  %.7f\n", velListX[p], velListY[p], velListZ[p]);
    }
}


inline void count_and_get_div(){
  num_cell_div = 0;
  hipMemcpy(cell_div, d_cell_div, No_of_C180s*sizeof(char), hipMemcpyDeviceToHost);
  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    if (cell_div[cellInd] == 1){
      cell_div[cellInd] = 0;
      cell_div_inds[num_cell_div] = cellInd;
      num_cell_div++;
    }
  }
  hipMemcpy(d_cell_div, cell_div, No_of_C180s*sizeof(char), hipMemcpyHostToDevice);
}



inline void calc_sys_CM(){ // Put this into a kernel at some point

  sysCMx = 0;
  sysCMy = 0;
  sysCMz = 0;

  for (int cellInd = 0; cellInd < No_of_C180s; cellInd++) {
    sysCMx += CMx[cellInd];
    sysCMy += CMy[cellInd];
    sysCMz += CMz[cellInd];
  }

  sysCMx = sysCMx/No_of_C180s;
  sysCMy = sysCMy/No_of_C180s;
  sysCMz = sysCMz/No_of_C180s;

}


inline float getRmax2(){
  float dx, dy, dz, Rmax2 = 0;
  for (int cell = 0; cell < No_of_C180s; cell++) {
    dx = CMx[cell] - sysCMx;
    dy = CMy[cell] - sysCMy;
    dz = CMz[cell] - sysCMz;

    Rmax2 = max(Rmax2, dx*dx + dy*dy + dz*dz);

  }

  return Rmax2;

}

inline int num_cells_far(){

  if (num_cell_div == 0 || No_of_C180s < 50) return 0;

  hipMemcpy(CMx, d_CMx, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(CMy, d_CMy, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(CMz, d_CMz, No_of_C180s*sizeof(float), hipMemcpyDeviceToHost);

  calc_sys_CM();

  float dx, dy, dz, dr2;
  float Rmax2 = getRmax2();
  int farCellCount = 0;

  for (int cell = No_of_C180s - num_cell_div; cell < No_of_C180s; cell++) { // Only check the newest cells
    dx = CMx[cell] - sysCMx;
    dy = CMy[cell] - sysCMy;
    dz = CMz[cell] - sysCMz;

    dr2 = dx*dx + dy*dy + dz*dz;

    if (dr2 > radFrac*radFrac*Rmax2)
      farCellCount++;
  }

  return farCellCount;

}
