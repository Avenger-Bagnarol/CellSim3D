#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "VectorFunctions.hpp"

__global__ void volumes( int No_of_C180s, int *C180_56,
                         float *X,    float *Y,   float *Z,
                         float *CMx , float *CMy, float *CMz, float *vol,
                         char* cell_div, float divVol, bool checkSphericity,
                         float* areaList){
    __shared__ float locX[192];
    __shared__ float locY[192];
    __shared__ float locZ[192];
    __shared__ float volume;
    __shared__ float volume2;
    __shared__ float area; 

    int fullerene = blockIdx.x;
    int tid       = threadIdx.x;

    if ( tid < 180 ){
        locX[tid] = X[192*fullerene+tid] -CMx[fullerene];
        locY[tid] = Y[192*fullerene+tid] -CMy[fullerene];
        locZ[tid] = Z[192*fullerene+tid] -CMz[fullerene];
    }

    if ( tid == 0){
        volume = 0.0f;
        volume2 = 0.0f; 
        area = 0.0f;
    }

    __syncthreads();

    if ( tid < 92 ){

        float avX = 0.0f;
        float avY = 0.0f;
        float avZ = 0.0f;

        for ( int i = 0; i < 5; ++i ){
            avX += locX[C180_56[7*tid+i]];
            avY += locY[C180_56[7*tid+i]];
            avZ += locZ[C180_56[7*tid+i]];
        }

        float avefactor = 0.166666667f;
        if ( tid < 12 )
        {
            avefactor = 0.2f;
        }
        else
        {
            avX += locX[C180_56[7*tid+5]];
            avY += locY[C180_56[7*tid+5]];
            avZ += locZ[C180_56[7*tid+5]];
        }

        avX *= avefactor;
        avY *= avefactor;
        avZ *= avefactor;

        float totvol = 0.0f;
        float totvol2 = 0.0f;
        float n1 = 0.0f;
        float n2 = 0.0f;
        float n3 = 0.0f;
        float faceArea = 0.0f;

        float face[7*3];

        float3 p0 = make_float3(avX, avY, avZ); 
        float3 p1, p2;

        for ( int i = 0; i < 6; ++i ){
            n1 = (locY[C180_56[7*tid+i+1]]*avZ-avY*locZ[C180_56[7*tid+i+1]])*locX[C180_56[7*tid+i]];
            n2 = (locZ[C180_56[7*tid+i+1]]*avX-avZ*locX[C180_56[7*tid+i+1]])*locY[C180_56[7*tid+i]];
            n3 = (locX[C180_56[7*tid+i+1]]*avY-avX*locY[C180_56[7*tid+i+1]])*locZ[C180_56[7*tid+i]];
            totvol += fabsf(n1+n2+n3);


            p1.x = locX[C180_56[7*tid+i]];
            p1.y = locY[C180_56[7*tid+i]];
            p1.z = locZ[C180_56[7*tid+i]];

            p2.x = locX[C180_56[7*tid+i+1]];
            p2.y = locY[C180_56[7*tid+i+1]];
            p2.z = locZ[C180_56[7*tid+i+1]];

            //totvol2 += dot(p0, cross(p1, p2)); 

            if (checkSphericity){
       
                // Get vectors that define a triangle 1, 2
                float x1 = locX[C180_56[7*tid+i]] - avX;
                float y1 = locY[C180_56[7*tid+i]] - avY;
                float z1 = locZ[C180_56[7*tid+i]] - avZ;

                float x2 = locX[C180_56[7*tid+i+1]] - avX;
                float y2 = locY[C180_56[7*tid+i+1]] - avY;
                float z2 = locZ[C180_56[7*tid+i+1]] - avZ;

                p1.x = p1.x - p0.x;
                p1.y = p1.y - p0.y;
                p1.z = p1.z - p0.z;

                p2.x = p2.x - p0.x;
                p2.y = p2.y - p0.y;
                p2.z = p2.z - p0.z;

                // now 1 will hold 1X2
                float xx = y1*z2 - z1*y2;
                float yy = z1*x2 - x1*z2;
                float zz = x1*y2 - y1*x2;

                // area of triangle is then 0.5*|1|
                faceArea += 0.5 * sqrt(xx*xx + yy*yy + zz*zz);
            }
        }
        atomicAdd(&volume, totvol);
        atomicAdd(&volume2, totvol2); 
    
        if (checkSphericity)
            atomicAdd(&area, faceArea); 
    }

    __syncthreads();

    if ( tid == 0){
        // if (isnan(vol[fullerene])){
        //     printf("OH SHIT: nan volume for cell %d", fullerene); 
        //     asm("trap;");
        // }

        
     
        volume = volume/6.0; 
        vol[fullerene] = volume;
        bool divide = 0; 
     
        if (volume > divVol){
            cell_div[fullerene] = 1;
            //printf("Cell %d volume =%f, volume2=%f \n", fullerene, volume/6.0, volume2/6.0); 
            //divide = 1; 
        }

        if (checkSphericity){
            areaList[fullerene] = area; 
            float psi = 4.835975862049408 * powf(volume, 2.0/3.0)/area;
            if ((1.0f - psi) > 0.1){
                cell_div[fullerene] = 0;
                //printf("cell %d division rejected\n", fullerene);
            }
        }
    }
    
}


__global__ void NewVolumes( int No_of_C180s, int *C180_56,
                            float *X,    float *Y,   float *Z,
                            float *CMx , float *CMy, float *CMz, float *voll,
                            char* cell_div, float divVol)
{
    __shared__ float locX[180];
    __shared__ float locY[180];
    __shared__ float locZ[180];
    __shared__ float volume;

    int blockInd = blockIdx.x;
    int cellInd = threadIdx.x;

    for (int i = 0; i < 180; i++){
        locX[i] = X[cellInd*192+i] - CMx[cellInd];
        locY[i] = Y[cellInd*192+i] - CMy[cellInd];
        locZ[i] = Z[cellInd*192+i] - CMz[cellInd];
    }
    
    float vol = 0.0f;
    float area = 0.0f;
    int N1, N2, N3, N4, N5, N6, N7;
    float CMFx, CMFy, CMFz;
    float b1x, b1y, b1z;
    float b2x, b2y, b2z;

    float volFace = 0.0f;
    float CPx, CPy, CPz; 
    
    for (int face = 0; face < 92; face++){
        N1 = C180_56[7*face+1];
        N2 = C180_56[7*face+2];
        N3 = C180_56[7*face+3];
        N4 = C180_56[7*face+4];
        N5 = C180_56[7*face+5];
        N6 = C180_56[7*face+6];
        
        CMFx = 1/6.0 * (locX[N1] + locX[N2] + locX[N3] +
                        locX[N4] + locX[N5] + locX[N6]);
        
        CMFy = 1/6.0 * (locY[N1] + locY[N2] + locY[N3] +
                        locY[N4] + locY[N5] + locY[N6]);
        
        CMFz = 1/6.0 * (locZ[N1] + locZ[N2] + locZ[N3] +
                        locZ[N4] + locZ[N5] + locZ[N6]);

        // each face makes a hex-prism (extruded hexagon)
        // the volume of which can be estimated by volume of 3
        // parallelipipeds
        // volume defined as u.(v x w) where u, v, w define length, width
        // height vectors of the piped.

        b1x = locX[N1] - locX[N2]; 
        b1y = locY[N1] - locY[N2]; 
        b1z = locZ[N1] - locZ[N2]; 

        b2x = locX[N3] - locX[N2];
        b2y = locY[N3] - locY[N2];
        b2z = locZ[N3] - locZ[N2];

        CPx = b1y*b2x - b1z*b2y;
        CPy = b1x*b2z - b1z*b2x;
        CPz = b1x*b2y - b1y*b2x;

        area += sqrt(CPx*CPx + CPy*CPy + CPz*CPz); 
        
        volFace += fabs(CMFx*CPx - CMFy*CPy + CMFz*CPz);
        
        
        b1x = locX[N3] - locX[N4]; 
        b1y = locY[N3] - locY[N4]; 
        b1z = locZ[N3] - locZ[N4]; 

        b2x = locX[N5] - locX[N4];
        b2y = locY[N5] - locY[N4];
        b2z = locZ[N5] - locZ[N4];
        
        CPx = b1y*b2x - b1z*b2y;
        CPy = b1x*b2z - b1z*b2x;
        CPz = b1x*b2y - b1y*b2x;

        area += sqrt(CPx*CPx + CPy*CPy + CPz*CPz); 
        
        volFace += fabs(CMFx*CPx - CMFy*CPy + CMFz*CPz);        
        

        b1x = locX[N5] - locX[N6]; 
        b1y = locY[N5] - locY[N6]; 
        b1z = locZ[N5] - locZ[N6]; 
        
        b2x = locX[N1] - locX[N6];
        b2y = locY[N1] - locY[N6];
        b2z = locZ[N1] - locZ[N6];
        
        CPx = b1y*b2x - b1z*b2y;
        CPy = b1x*b2z - b1z*b2x;
        CPz = b1x*b2y - b1y*b2x;

        area += sqrt(CPx*CPx + CPy*CPy + CPz*CPz); 
        
        volFace += fabs(CMFx*CPx - CMFy*CPy + CMFz*CPz);
        
        vol += volFace;
    }

    
}
