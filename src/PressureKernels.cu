#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "postscript.h"

__global__ void PressureUpdate (float* d_pressList, float minPressure,
                                float maxPressure, float inc, int No_of_C180s){

    int cellInd = blockIdx.x*blockDim.x + threadIdx.x;
    float pressure = d_pressList[cellInd];
    
    if (cellInd < No_of_C180s){
        if (d_pressList[cellInd] < maxPressure){
            d_pressList[cellInd] += inc;
        }
    }
    
}


__global__ void PressureReset (int* d_resetIndices, float* d_pressList,
                               float pressureValue, int numCells){

    // d_resetIndices is an array of indeces to cells that have to have their
    // pressures reset

    int setInd = blockIdx.x*blockDim.x + threadIdx.x;
    if (setInd < numCells){
        // if (!d_resetIndices[setInd]){
        //     printf("WARNING:Trying to set pressure  of wrong cell\n");
        //     printf("Invalid cell index: %d setId: %d  numCells: %d", d_resetIndices[setInd], setInd, numCells); 
        // }

        d_pressList[d_resetIndices[setInd]] = pressureValue;
        d_resetIndices[setInd] = -1;
    }
}
